#include "hip/hip_runtime.h"
#include "opp_cuda.h"

/*******************************************************************************/
__global__ void export_halo_gather(int *list, char *dat, int copy_size,
                                   int elem_size, char *export_buffer) 
{
    const int id = OPP_DEVICE_GLOBAL_LINEAR_ID;

    if (id < copy_size) {
        int off = 0;
        if (elem_size % 16 == 0) {
            const int elem_size_over_16 = (elem_size / 16);
            off += elem_size;
            for (int i = 0; i < elem_size_over_16; i++) {
                ((double2 *)(export_buffer + id * elem_size))[i] =
                    ((double2 *)(dat + list[id] * elem_size))[i];
            }
        } 
        else if (elem_size % 8 == 0) {
            const int elem_size_over_8 = (elem_size / 8);
            off += elem_size;
            for (int i = 0; i < elem_size_over_8; i++) {
                ((double *)(export_buffer + id * elem_size))[i] =
                    ((double *)(dat + list[id] * elem_size))[i];
            }
        }
        for (int i = off; i < elem_size; i++) {
            export_buffer[id * elem_size + i] = dat[list[id] * elem_size + i];
        }
    }
}
__global__ void export_halo_gather_soa(int *list, char *dat, int copy_size,
                        int elem_size, char *export_buffer, int set_size, int dim) 
{
    const int id = OPP_DEVICE_GLOBAL_LINEAR_ID;
    const int size_of = elem_size / dim;
    
    if (id < copy_size) {
        if (size_of == 8) {
            for (int i = 0; i < dim; i++) {
                ((double *)(export_buffer + id * elem_size))[i] =
                    ((double *)(dat + list[id] * size_of))[i * set_size];
            }
        } 
        else {
            for (int i = 0; i < dim; i++) {
                for (int j = 0; j < size_of; j++) {
                    export_buffer[id * elem_size + i * size_of + j] =
                        dat[list[id] * size_of + i * set_size * size_of + j];
                }
            }
        }
    }
}

void gather_data_to_buffer(opp_arg arg, halo_list exp_exec_list, halo_list exp_nonexec_list) 
{
    const int blocks = 1 + ((exp_exec_list->size - 1) / opp_const_threads_per_block);
    const int blocks2 = 1 + ((exp_nonexec_list->size - 1) / opp_const_threads_per_block);

    if (strstr(arg.dat->type, ":soa") != NULL || (OPP_auto_soa && arg.dat->dim > 1)) {
        const int set_size = arg.dat->set->size + arg.dat->set->exec_size +
                    arg.dat->set->nonexec_size;

        export_halo_gather_soa<<<blocks, opp_const_threads_per_block>>>(
            export_exec_list_d[arg.dat->set->index], arg.data_d,
            exp_exec_list->size, arg.dat->size, arg.dat->buffer_d, set_size,
            arg.dat->dim);

        export_halo_gather_soa<<<blocks2, opp_const_threads_per_block>>>(
            export_nonexec_list_d[arg.dat->set->index], arg.data_d,
            exp_nonexec_list->size, arg.dat->size,
            arg.dat->buffer_d + exp_exec_list->size * arg.dat->size, set_size,
            arg.dat->dim);

    } 
    else {
        export_halo_gather<<<blocks, opp_const_threads_per_block>>>(
            export_exec_list_d[arg.dat->set->index], arg.data_d,
            exp_exec_list->size, arg.dat->size, arg.dat->buffer_d);

        export_halo_gather<<<blocks2, opp_const_threads_per_block>>>(
            export_nonexec_list_d[arg.dat->set->index], arg.data_d,
            exp_nonexec_list->size, arg.dat->size,
            arg.dat->buffer_d + exp_exec_list->size * arg.dat->size);
    }

    OPP_DEVICE_SYNCHRONIZE();
}

/*******************************************************************************/
__global__ void import_halo_scatter_soa(int offset, char *dat, int copy_size,
                        int elem_size, char *import_buffer, int set_size, int dim) 
{
    const int id = OPP_DEVICE_GLOBAL_LINEAR_ID;
    const int size_of = elem_size / dim;
    
    if (id < copy_size) {
        if (size_of == 8) {
            for (int i = 0; i < dim; i++) {
                ((double *)(dat + (offset + id) * size_of))[i * set_size] =
                    ((double *)(import_buffer + id * elem_size))[i];
            }
        } 
        else {
            for (int i = 0; i < dim; i++) {
                for (int j = 0; j < size_of; j++) {
                    dat[(offset + id) * size_of + i * set_size * size_of + j] =
                        import_buffer[id * elem_size + i * size_of + j];
                }
            }
        }
    }
}

void scatter_data_from_buffer(opp_arg arg) 
{
    const int blocks = 1 + ((arg.dat->set->exec_size - 1) / opp_const_threads_per_block);
    const int blocks2 = 1 + ((arg.dat->set->nonexec_size - 1) / opp_const_threads_per_block);

    if (strstr(arg.dat->type, ":soa") != NULL || (OPP_auto_soa && arg.dat->dim > 1)) {
        const int set_size = arg.dat->set->size + arg.dat->set->exec_size +
                    arg.dat->set->nonexec_size;
        int offset = arg.dat->set->size;
        int copy_size = arg.dat->set->exec_size;

        import_halo_scatter_soa<<<blocks, opp_const_threads_per_block>>>(
            offset, arg.data_d, copy_size, arg.dat->size, arg.dat->buffer_d_r,
            set_size, arg.dat->dim);

        offset += arg.dat->set->exec_size;
        copy_size = arg.dat->set->nonexec_size;

        import_halo_scatter_soa<<<blocks2, opp_const_threads_per_block>>>(
            offset, arg.data_d, copy_size, arg.dat->size,
            arg.dat->buffer_d_r + arg.dat->set->exec_size * arg.dat->size, set_size,
            arg.dat->dim);
    }
}

/*******************************************************************************/