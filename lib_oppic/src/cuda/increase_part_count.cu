#include "hip/hip_runtime.h"

/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <oppic_cuda.h>

//*************************************************************************************************
__global__ void opp_cuda_AssignMeshRelation(
    int *__restrict mesh_relation,
    const int *__restrict distribution,
    int start,
    int end,
    int inj_start,
    int inlet_size
    ) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {    
        int n = tid + start;

        for (int i = 0; i < inlet_size; i++)
        {
            if (tid < distribution[i])
            {
                // assign inlet face index as the injected particle mesh relation
                mesh_relation[n + inj_start] = i; 
                break;
            } 
        }  
    }
}


//****************************************
void opp_inc_part_count_with_distribution(opp_set particles_set, 
    int num_particles_to_insert, opp_dat iface_dist)
{
    if (OP_DEBUG) opp_printf("opp_inc_part_count_with_distribution", "num_particles_to_insert [%d]", 
        num_particles_to_insert);

    opp_profiler->start("IncPartCountWithDistribution");

    opp_dat mesh_rel_dat  = particles_set->mesh_relation_dat;

    // int nargs = 1;
    // opp_arg args[nargs];
    // args[0] = opp_get_arg(mesh_rel_dat, OP_READ);

    // int set_size = opp_mpi_halo_exchanges_grouped(particles_set, nargs, args, Device_CPU);

    // TODO : BUG What happens if the complete particle is dirty in device?

    oppic_increase_particle_count(particles_set, num_particles_to_insert);

    int nargs1 = 2;
    opp_arg args1[nargs1];

    // if iface particle distribution is dirty in device, get it to the device
    args1[0] = opp_get_arg(iface_dist, OP_READ);
    args1[1] = opp_get_arg(mesh_rel_dat, OP_WRITE);

    int set_size = opp_mpi_halo_exchanges_grouped(particles_set, nargs1, args1, Device_GPU);
    if (set_size > 0) 
    {
        int start     = 0;
        int end       = particles_set->diff;
        int inj_start = (particles_set->size - particles_set->diff);

        if (end - start > 0) 
        {
            int nthread = OPP_GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;

            opp_cuda_AssignMeshRelation<<<nblocks, nthread>>>(
                (int *) mesh_rel_dat->data_d,
                (int *) iface_dist->data_d,
                start, 
                end, 
                inj_start,
                iface_dist->set->size);
        }
    }

    opp_mpi_set_dirtybit_grouped(nargs1, args1, Device_GPU);
    cutilSafeCall(hipDeviceSynchronize());

    opp_profiler->end("IncPartCountWithDistribution");
}

