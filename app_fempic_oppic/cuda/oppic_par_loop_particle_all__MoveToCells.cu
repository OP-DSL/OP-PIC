#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

int opDat0_MoveToCells_all_stride_OPPIC_HOST = -1;
int opDat1_MoveToCells_all_stride_OPPIC_HOST = -1;
int opDat4_MoveToCells_all_stride_OPPIC_HOST = -1;
int opDat5_MoveToCells_all_stride_OPPIC_HOST = -1;

__constant__ int opDat0_MoveToCells_all_stride_OPPIC_CONSTANT;
__constant__ int opDat1_MoveToCells_all_stride_OPPIC_CONSTANT;
__constant__ int opDat4_MoveToCells_all_stride_OPPIC_CONSTANT;
__constant__ int opDat5_MoveToCells_all_stride_OPPIC_CONSTANT;


//user function
//*************************************************************************************************
__device__ void move_all_particles_to_cell__kernel(
    int* move_status,
    const double* part_pos,
    double* part_lc,
    int* current_cell_index,
    const double *cell_volume,
    const double *cell_det,
    const int *cell_connectivity,
    const bool* search)
{
    bool inside = true;

    for (int i=0; i < NODES_PER_CELL; i++) /*loop over vertices*/
    {
        part_lc[i * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT] = (1.0/6.0) * (
            cell_det[(0 + i * NODES_PER_CELL) * opDat4_MoveToCells_all_stride_OPPIC_CONSTANT] - 
            cell_det[(1 + i * NODES_PER_CELL) * opDat4_MoveToCells_all_stride_OPPIC_CONSTANT] * part_pos[0 * opDat0_MoveToCells_all_stride_OPPIC_CONSTANT] + 
            cell_det[(2 + i * NODES_PER_CELL) * opDat4_MoveToCells_all_stride_OPPIC_CONSTANT] * part_pos[1 * opDat0_MoveToCells_all_stride_OPPIC_CONSTANT] - 
            cell_det[(3 + i * NODES_PER_CELL) * opDat4_MoveToCells_all_stride_OPPIC_CONSTANT] * part_pos[2 * opDat0_MoveToCells_all_stride_OPPIC_CONSTANT]
                ) / (*cell_volume);
        
        if (part_lc[i * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT] < 0 || part_lc[i * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT] > 1.0) inside = false;
    }

    if (inside)
    {
        *move_status = MOVE_DONE;
        return;
    }

    if (*search) 
    {
        (*current_cell_index)++; // outside the last known cell, Increment the cell_index to search in the full mesh
        return;
    }

    // outside the last known cell, find most negative weight and use that cell_index to reduce computations
    int min_i = 0;
    double min_lc = part_lc[0 * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT];
    
    for (int i=1; i<NEIGHBOUR_CELLS; i++)
    {
        if (part_lc[i * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT] < min_lc) 
        {
            min_lc = part_lc[i * opDat1_MoveToCells_all_stride_OPPIC_CONSTANT];
            min_i = i;
        }
    }

    if (cell_connectivity[min_i * opDat5_MoveToCells_all_stride_OPPIC_CONSTANT] >= 0) // is there a neighbor in this direction?
    {
        (*current_cell_index) = cell_connectivity[min_i * opDat5_MoveToCells_all_stride_OPPIC_CONSTANT];
        *move_status = NEED_MOVE;
    }
    else
    {
        (*current_cell_index) = MAX_CELL_INDEX;
        *move_status = NEED_REMOVE;
    }
}


// CUDA kernel function
//*************************************************************************************************
__global__ void oppic_cuda_all_MoveToCells(
    int *__restrict d_cell_index,
    double *__restrict dir_arg0,
    double *__restrict dir_arg1,
    int *__restrict dir_arg2,
    const double *__restrict ind_arg3,
    const double *__restrict ind_arg4,
    const int *__restrict ind_arg5,
    const bool *__restrict dir_arg6,
    int *__restrict particle_statuses,
    int start,
    int end,
    int num_cells) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;
        
        int& map0idx = dir_arg2[n];
        // int& map0idx = d_cell_index[n]; // TODO : I dont know why this isn't working ??? dir_arg2 and d_cell_index has same pointer values, but this get stuck!

        int move_status = (int)NEED_MOVE;

        do
        {
            //user-supplied kernel call
            move_all_particles_to_cell__kernel(
                &(move_status),
                (dir_arg0 + n),         // part_pos
                (dir_arg1 + n),         // part_weights
                (dir_arg2 + n),         // part_cell_index
                (ind_arg3 + map0idx),   // cell_volume
                (ind_arg4 + map0idx),   // cell_det
                (ind_arg5 + map0idx),   // cell_connectivity
                dir_arg6                // full_mesh_search
            );                
            
        } while ((move_status == (int)NEED_MOVE) && (map0idx < num_cells));

        if (move_status == (int)NEED_REMOVE) /*outside the mesh*/
        {    
            printf("***** move_all_particles_to_cell__kernel : Need to remove particle %d\n", n);
            particle_statuses[n] = (int)NEED_REMOVE;
        }
        else if (move_status != MOVE_DONE)
        {
            printf("ERROR *********** move_all_particles_to_cell__kernel returned status[%d] for particle [%d]\n", move_status, n);
        }
    }
}

//*************************************************************************************************
void oppic_par_loop_particle_all__MoveToCells(
    oppic_set set,      // particles_set
    oppic_arg arg0,     // part_position,
    oppic_arg arg1,     // part_weights,
    oppic_arg arg2,     // part_cell_index,
    oppic_arg arg3,     // cell_volume,
    oppic_arg arg4,     // cell_det,
    oppic_arg arg5,     // cell_connectivity_map,
    oppic_arg arg6      // particles_injected       
    )
{ TRACE_ME;
    
    if (OP_DEBUG) printf("FEMPIC - oppic_par_loop_particle_all__MoveToCells num_particles %d\n", set->size);

    int nargs = 7;
    oppic_arg args[nargs];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;

    cutilSafeCall(hipMalloc(&(arg6.data_d), arg6.size));
    cutilSafeCall(hipMemcpy(arg6.data_d, arg6.data, arg6.size, hipMemcpyHostToDevice));

    oppic_init_particle_move(set);

    int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, Device_GPU);
    if (set_size > 0) 
    {
        opDat0_MoveToCells_all_stride_OPPIC_HOST = arg0.dat->set->size;
        opDat1_MoveToCells_all_stride_OPPIC_HOST = arg1.dat->set->size;
        opDat4_MoveToCells_all_stride_OPPIC_HOST = arg4.dat->set->size;
        opDat5_MoveToCells_all_stride_OPPIC_HOST = arg5.map->from->size;

        hipMemcpyToSymbol(HIP_SYMBOL(opDat0_MoveToCells_all_stride_OPPIC_CONSTANT), &opDat0_MoveToCells_all_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat1_MoveToCells_all_stride_OPPIC_CONSTANT), &opDat1_MoveToCells_all_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat4_MoveToCells_all_stride_OPPIC_CONSTANT), &opDat4_MoveToCells_all_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat5_MoveToCells_all_stride_OPPIC_CONSTANT), &opDat5_MoveToCells_all_stride_OPPIC_HOST, sizeof(int));

        int start   = 0;
        int end     = set->size;

        if (end - start > 0) 
        {
            int nthread = GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;

            oppic_cuda_all_MoveToCells<<<nblocks, nthread>>>(
                (int *)     set->cell_index_dat->data_d,
                (double *)  arg0.data_d,
                (double *)  arg1.data_d,
                (int *)     arg2.data_d,
                (double *)  arg3.data_d,
                (double *)  arg4.data_d,
                (int *)     arg5.data_d,
                (bool *)    arg6.data_d,
                (int *)     set->particle_statuses_d,
                start, 
                end, 
                set->cells_set->size);
        }
    }

    cutilSafeCall(hipDeviceSynchronize());

    oppic_finalize_particle_move(set);

    cutilSafeCall(hipFree(arg6.data_d));
    op_mpi_set_dirtybit_cuda(nargs, args);
    cutilSafeCall(hipDeviceSynchronize());
}

//*************************************************************************************************