
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <oppic_cuda.h>

// This assumes all the device data to be valid
void particle_sort_cuda(oppic_set set)
{ TRACE_ME;

    int set_capacity = set->set_capacity;
    int set_size_plus_removed = set->size + set->particle_remove_count;

    if (OP_DEBUG) printf("\tparticle_sort_cuda set [%s] with set capacity [%d] set size plus removed [%d]\n", set->name, set_capacity, set_size_plus_removed);

    thrust::device_ptr<int> cellIdx_dp = thrust::device_pointer_cast((int*)set->cell_index_dat->data_d);
    thrust::device_vector<int> cellIdx_dv(cellIdx_dp, cellIdx_dp + set_size_plus_removed);

    thrust::device_vector<int> i_dv(set_size_plus_removed);
    thrust::sequence(i_dv.begin(), i_dv.end());

    thrust::sort_by_key(cellIdx_dv.begin(), cellIdx_dv.end(), i_dv.begin());

    for (int i = 0; i < (int)set->particle_dats->size(); i++)
    {    
        oppic_dat& dat = set->particle_dats->at(i);

        if (!(strstr(dat->type, ":soa") != NULL || OP_auto_soa || (dat->dim > 1)))
        {
            std::cerr << "particle_sort_cuda not implemented for non SOA data structures [dat " << dat->name << "]" << std::endl;
        }

        if (strcmp(dat->type, "int") == 0)
        {
            sort_dat_according_to_index<int>(dat, i_dv, set_capacity, set_size_plus_removed);
        }
        else if (strcmp(dat->type, "double") == 0)
        {
            sort_dat_according_to_index<double>(dat, i_dv, set_capacity, set_size_plus_removed);
        }
        else
        {
            std::cerr << "particle_sort_cuda not implemented for data type " << dat->type << " [dat " << dat->name << "]" << std::endl;
        }
    }

    cutilSafeCall(hipDeviceSynchronize());
}
