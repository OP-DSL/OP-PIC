#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

__constant__ int opDat0_EnrichVelocity_stride_OPPIC_CONSTANT;
__constant__ int opDat1_EnrichVelocity_stride_OPPIC_CONSTANT;

int opDat0_EnrichVelocity_stride_OPPIC_HOST =- 1;
int opDat1_EnrichVelocity_stride_OPPIC_HOST =- 1;


//user function
//*************************************************************************************************
__device__ void enrich_velocity__kernel_gpu(
    double *vel,
    const double *cell_ef,
    const double *dt
)
{
    vel[0 * opDat0_EnrichVelocity_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[0 * opDat1_EnrichVelocity_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
    
    vel[1 * opDat0_EnrichVelocity_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[1 * opDat1_EnrichVelocity_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
    
    vel[2 * opDat0_EnrichVelocity_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[2 * opDat1_EnrichVelocity_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
}


// CUDA kernel function
//*************************************************************************************************
__global__ void oppic_cuda_EnrichVelocity(
    const int *__restrict d_cell_index,
    double *__restrict dir_arg0,
    const double *__restrict ind_arg1,
    const double *__restrict dir_arg2,
    int start,
    int end,
    int set_size
    ) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;
        
        int map0idx = d_cell_index[n];

        //user-supplied kernel call
        enrich_velocity__kernel_gpu(
            (dir_arg0 + n),
            (ind_arg1 + map0idx),
            dir_arg2
        );
    }
}


//*************************************************************************************************
void oppic_par_loop_inject__EnrichVelocity(
    oppic_set set,     // particles_set
    oppic_arg arg0,    // part_velocity,
    oppic_arg arg1,    // cell_electric_field,
    oppic_arg arg2     // const dt,    
    )
{ TRACE_ME;
    
    if (OP_DEBUG) printf("FEMPIC - oppic_par_loop_inject__EnrichVelocity num_particles %d\n", set->size);

    int nargs = 3;
    oppic_arg args[3];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;

    cutilSafeCall(hipMalloc(&(arg2.data_d), arg2.size));
    cutilSafeCall(hipMemcpy(arg2.data_d, arg2.data, arg2.size, hipMemcpyHostToDevice));

    int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, Device_GPU);
    if (set_size > 0) 
    {
        opDat0_EnrichVelocity_stride_OPPIC_HOST = arg0.dat->set->size;
        opDat1_EnrichVelocity_stride_OPPIC_HOST = arg1.dat->set->size;
        
        hipMemcpyToSymbol(HIP_SYMBOL(opDat0_EnrichVelocity_stride_OPPIC_CONSTANT), &opDat0_EnrichVelocity_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat1_EnrichVelocity_stride_OPPIC_CONSTANT), &opDat1_EnrichVelocity_stride_OPPIC_HOST, sizeof(int));

        int start = (set->size - set->diff);
        int end   = set->size;

        if (end - start > 0) 
        {
            int nthread = GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;
printf("FEMPIC - oppic_par_loop_inject__EnrichVelocity set_size %d %d %d %d *********************************************\n", set_size, start, end, nblocks);
            oppic_cuda_EnrichVelocity <<<nblocks, nthread>>> (
                (int *)     set->cell_index_dat->data_d,
                (double *)  arg0.data_d,
                (double *)  arg1.data_d,
                (double *)  arg2.data_d,
                start, 
                end, 
                set->size);
        }
    }
    
    cutilSafeCall(hipFree(arg2.data_d));
    op_mpi_set_dirtybit_cuda(nargs, args);
    cutilSafeCall(hipDeviceSynchronize());

// oppic_download_particle_set(set);
}

//*************************************************************************************************