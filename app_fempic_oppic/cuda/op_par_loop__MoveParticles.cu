#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

__constant__ int opDat0_MoveParticles_stride_OPPIC_CONSTANT;
__constant__ int opDat1_MoveParticles_stride_OPPIC_CONSTANT;
__constant__ int opDat2_MoveParticles_stride_OPPIC_CONSTANT;

int opDat0_MoveParticles_stride_OPPIC_HOST =- 1;
int opDat1_MoveParticles_stride_OPPIC_HOST =- 1;
int opDat2_MoveParticles_stride_OPPIC_HOST =- 1;

//user function
//*************************************************************************************************
__device__ void move_particles__kernel_gpu(
    double *pos,    
    double *vel,
    const double *ef,
    const double *dt
)
{
    vel[0 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] += (OP_CONST_charge / OP_CONST_mass * 
                                                            ef[0 * opDat2_MoveParticles_stride_OPPIC_CONSTANT] * (*dt));
    vel[1 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] += (OP_CONST_charge / OP_CONST_mass * 
                                                            ef[1 * opDat2_MoveParticles_stride_OPPIC_CONSTANT] * (*dt));
    vel[2 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] += (OP_CONST_charge / OP_CONST_mass * 
                                                            ef[2 * opDat2_MoveParticles_stride_OPPIC_CONSTANT] * (*dt));
    
    pos[0 * opDat0_MoveParticles_stride_OPPIC_CONSTANT] += vel[0 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] * (*dt); // v = u + at
    
    pos[1 * opDat0_MoveParticles_stride_OPPIC_CONSTANT] += vel[1 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] * (*dt); // v = u + at
    
    pos[2 * opDat0_MoveParticles_stride_OPPIC_CONSTANT] += vel[2 * opDat1_MoveParticles_stride_OPPIC_CONSTANT] * (*dt); // v = u + at
}


// CUDA kernel function
//*************************************************************************************************
__global__ void op_cuda_MoveParticles(
    double *__restrict dir_arg0,
    double *__restrict dir_arg1,
    const double *__restrict dir_arg2,
    const double *__restrict dir_arg3,
    int start,
    int end,
    int set_size
    ) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;

        //user-supplied kernel call
        move_particles__kernel_gpu(
            (dir_arg0 + n),
            (dir_arg1 + n),
            (dir_arg2 + n),
            dir_arg3
        );
    }
}


//*************************************************************************************************
void op_par_loop_all__MoveParticles(
    op_set set,     // particles_set
    op_arg arg0,    // part_position,
    op_arg arg1,    // part_velocity,
    op_arg arg2,    // part_electric_field,
    op_arg arg3     // const dt 
    )
{ TRACE_ME;
    
    if (OP_DEBUG) printf("FEMPIC - op_par_loop_all__MoveParticles num_particles %d\n", set->size);

    opDat0_MoveParticles_stride_OPPIC_HOST = arg0.dat->set->size;
    opDat1_MoveParticles_stride_OPPIC_HOST = arg1.dat->set->size;
    opDat2_MoveParticles_stride_OPPIC_HOST = arg2.dat->set->size;

    hipMemcpyToSymbol(HIP_SYMBOL(opDat0_MoveParticles_stride_OPPIC_CONSTANT), &opDat0_MoveParticles_stride_OPPIC_HOST, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(opDat1_MoveParticles_stride_OPPIC_CONSTANT), &opDat1_MoveParticles_stride_OPPIC_HOST, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(opDat2_MoveParticles_stride_OPPIC_CONSTANT), &opDat2_MoveParticles_stride_OPPIC_HOST, sizeof(int));

    int start = 0;
    int end   = set->size;

    if (end - start > 0) 
    {
        int nthread = GPU_THREADS_PER_BLOCK;
        int nblocks = (end - start - 1) / nthread + 1;

        op_cuda_MoveParticles <<<nblocks, nthread>>> (
            (double *)  arg0.data_d,
            (double *)  arg1.data_d,
            (double *)  arg2.data_d,
            (double *)  arg3.data_d,
            start, 
            end, 
            set->size);
    }
}

//*************************************************************************************************