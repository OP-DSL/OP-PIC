
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <oppic_cuda.h>


void particle_sort_cuda(oppic_set set)
{ TRACE_ME;

    if (OP_DEBUG) printf("particle_sort_cuda set [%s]\n", set->name);

    int set_size = set->size;

    thrust::device_ptr<int> cellIdx_dp = thrust::device_pointer_cast((int*)set->cell_index_dat->data_d);
    thrust::device_vector<int> cellIdx_dv(cellIdx_dp, cellIdx_dp + set_size);

    thrust::device_vector<int> i_dv(set_size);
    thrust::sequence(i_dv.begin(), i_dv.end());

    thrust::sort_by_key(cellIdx_dv.begin(), cellIdx_dv.end(), i_dv.begin());

    for (int i = 0; i < (int)set->particle_dats->size(); i++)
    {    
        auto& dat = set->particle_dats->at(i);

        if (!(strstr(dat->type, ":soa") != NULL || OP_auto_soa || (dat->dim > 1)))
        {
            std::cerr << "particle_sort_cuda not implemented for non SOA data structures [dat " << dat->name << "]" << std::endl;
        }

        if (strcmp(dat->type, "int") == 0)
        {
            sort_dat_according_to_index<int>(dat, i_dv, set_size);
        }
        else if (strcmp(dat->type, "double") == 0)
        {
            sort_dat_according_to_index<double>(dat, i_dv, set_size);
        }
        else
        {
            std::cerr << "particle_sort_cuda not implemented for data type " << dat->type << " [dat " << dat->name << "]" << std::endl;
        }
    }

    cutilSafeCall(hipDeviceSynchronize());
}


// void particle_sort_cuda(oppic_set set)
// { TRACE_ME;

//     if (OP_DEBUG) printf("particle_sort_cuda set [%s]\n", set->name);

//     int set_size = set->size;
// printf("XXXXX set_size %d\n", set_size);
//     thrust::device_ptr<int> cellIdx_dp = thrust::device_pointer_cast((int*)set->cell_index_dat->data_d);
//     thrust::device_vector<int> cellIdx_dv(cellIdx_dp, cellIdx_dp + set_size);

//     thrust::device_vector<int> i_dv(set_size);
//     thrust::sequence(i_dv.begin(), i_dv.end());

// thrust::host_vector<int> h_cellIdx_dv = cellIdx_dv;
// thrust::host_vector<int> h_i_dv = i_dv;

// std::cout << "h_cellIdx_dv: " ;
// thrust::copy_n(h_cellIdx_dv.begin(), set_size, std::ostream_iterator<int>(std::cout, ","));
// std::cout << std::endl << "h_i_dv: " ;
// thrust::copy_n(h_i_dv.begin(), set_size, std::ostream_iterator<int>(std::cout, ","));
// std::cout << std::endl;

//     thrust::sort_by_key(cellIdx_dv.begin(), cellIdx_dv.end(), i_dv.begin());

// thrust::host_vector<int> h_i_dv1 = i_dv;
// std::cout << "i_dv SORTED: " ;
// thrust::copy_n(h_i_dv1.begin(), set_size, std::ostream_iterator<int>(std::cout, ","));
// std::cout << std::endl;

//     for (int i = 0; i < (int)set->particle_dats->size(); i++) // TODO: Remove this -1 to sort cell index
//     {    
//         auto& dat = set->particle_dats->at(i);

//         if (!(strstr(dat->type, ":soa") != NULL || OP_auto_soa || (dat->dim > 1)))
//         {
//             std::cerr << "particle_sort_cuda not implemented for non SOA data structures [dat " << dat->name << "]" << std::endl;
//         }

//         if (strcmp(dat->type, "int") == 0)
//         {
//             sort_dat_according_to_index<int>(dat, i_dv, set_size);
//         }
//         else if (strcmp(dat->type, "double") == 0)
//         {
//             sort_dat_according_to_index<double>(dat, i_dv, set_size);
//         }
//         else
//         {
//             std::cerr << "particle_sort_cuda not implemented for data type " << dat->type << " [dat " << dat->name << "]" << std::endl;
//         }
//     }

// thrust::device_ptr<int> cellIdx_dp1 = thrust::device_pointer_cast((int*)set->cell_index_dat->data_d);
// thrust::device_vector<int> cellIdx_dv1(cellIdx_dp1, cellIdx_dp1 + set_size);
// thrust::host_vector<int> h_i_dv11 = cellIdx_dv1;
// std::cout << "SORTED CELL IDX SORTED: " ;
// thrust::copy_n(h_i_dv11.begin(), set_size, std::ostream_iterator<int>(std::cout, ","));
// std::cout << std::endl;

//     cutilSafeCall(hipDeviceSynchronize());
// }