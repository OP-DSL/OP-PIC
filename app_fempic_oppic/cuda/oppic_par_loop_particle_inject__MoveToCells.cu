#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

int opDat0_MoveToCells_inject_stride_OPPIC_HOST = -1;
int opDat1_MoveToCells_inject_stride_OPPIC_HOST = -1;
int opDat3_MoveToCells_inject_stride_OPPIC_HOST = -1;
int opDat5_MoveToCells_inject_stride_OPPIC_HOST = -1;
int opDat6_MoveToCells_inject_stride_OPPIC_HOST = -1;
int opDat7_MoveToCells_inject_stride_OPPIC_HOST = -1;

__constant__ int opDat0_MoveToCells_inject_stride_OPPIC_CONSTANT;
__constant__ int opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT;
__constant__ int opDat3_MoveToCells_inject_stride_OPPIC_CONSTANT;
__constant__ int opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT;
__constant__ int opDat6_MoveToCells_inject_stride_OPPIC_CONSTANT;
__constant__ int opDat7_MoveToCells_inject_stride_OPPIC_CONSTANT;


//user function
//*************************************************************************************************
__device__ void move_injected_particles_to_cell__kernel(
    int* move_status,
    const double* part_pos,
    double* part_lc,
    int* current_cell_index,
    double* part_vel,
    const double *cell_volume,
    const double *cell_det,
    const double *cell_ef,
    const int *cell_connectivity,
    const bool* search,
    const double* dt)
{
    bool inside = true;

    for (int i=0; i < NODES_PER_CELL; i++) /*loop over vertices*/
    {
        part_lc[i * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT] = (1.0/6.0) * (
            cell_det[(0 + i * NODES_PER_CELL) * opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT] - 
            cell_det[(1 + i * NODES_PER_CELL) * opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT] * part_pos[0 * opDat0_MoveToCells_inject_stride_OPPIC_CONSTANT] + 
            cell_det[(2 + i * NODES_PER_CELL) * opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT] * part_pos[1 * opDat0_MoveToCells_inject_stride_OPPIC_CONSTANT] - 
            cell_det[(3 + i * NODES_PER_CELL) * opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT] * part_pos[2 * opDat0_MoveToCells_inject_stride_OPPIC_CONSTANT]
                ) / (*cell_volume);
        
        if (part_lc[i * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT] < 0 || part_lc[i * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT] > 1.0) inside = false;
    }

    if (inside)
    {
        *move_status = MOVE_DONE;
        
        part_vel[0 * opDat3_MoveToCells_inject_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[0 * opDat6_MoveToCells_inject_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
    
        part_vel[1 * opDat3_MoveToCells_inject_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[1 * opDat6_MoveToCells_inject_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
    
        part_vel[2 * opDat3_MoveToCells_inject_stride_OPPIC_CONSTANT] -= OP_CONST_CUDA_charge / OP_CONST_CUDA_mass * 
                                                        cell_ef[2 * opDat6_MoveToCells_inject_stride_OPPIC_CONSTANT] * (0.5 * (*dt));
        return;
    }

    if (*search) 
    {
        (*current_cell_index)++; // outside the last known cell, Increment the cell_index to search in the full mesh
        return;
    }

    // outside the last known cell, find most negative weight and use that cell_index to reduce computations
    int min_i = 0;
    double min_lc = part_lc[0 * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT];
    
    for (int i=1; i<NEIGHBOUR_CELLS; i++)
    {
        if (part_lc[i * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT] < min_lc) 
        {
            min_lc = part_lc[i * opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT];
            min_i = i;
        }
    }

    if (cell_connectivity[min_i * opDat7_MoveToCells_inject_stride_OPPIC_CONSTANT] >= 0) // is there a neighbor in this direction?
    {
        (*current_cell_index) = cell_connectivity[min_i * opDat7_MoveToCells_inject_stride_OPPIC_CONSTANT];
        *move_status = NEED_MOVE;
    }
    else
    {
        (*current_cell_index) = MAX_CELL_INDEX;
        *move_status = NEED_REMOVE;
    }
}


// CUDA kernel function
//*************************************************************************************************
__global__ void oppic_cuda_injected_MoveToCells(
    int *__restrict d_cell_index,
    double *__restrict dir_arg0,            // part_position,
    double *__restrict dir_arg1,            // part_weights,
    int *__restrict dir_arg2,               // part_cell_index,
    double *__restrict ind_arg3,            // part_velocity,
    const double *__restrict ind_arg4,      // cell_volume,
    const double *__restrict ind_arg5,      // cell_det,
    const double *__restrict ind_arg6,      // cell_electric_field,
    const int *__restrict ind_arg7,         // cell_connectivity_map,
    const bool *__restrict dir_arg8,        // particles_injected,
    const double *__restrict dir_arg9,      // dt,
    int *__restrict particle_statuses,      // mark particles as MOVE_DONE, NEED_REMOVE etc...
    int start,
    int end,
    int num_cells) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;
        
        int& map0idx = dir_arg2[n];
        // int& map0idx = d_cell_index[n]; // TODO : I dont know why this isn't working ??? dir_arg2 and d_cell_index has same pointer values, but this get stuck!

        int move_status = (int)NEED_MOVE;

        do
        {
            //user-supplied kernel call
            move_injected_particles_to_cell__kernel(
                &(move_status),
                (dir_arg0 + n),         // part_pos
                (dir_arg1 + n),         // part_weights
                (dir_arg2 + n),         // part_cell_index
                (ind_arg3 + n),         // part_velocity,
                (ind_arg4 + map0idx),   // cell_volume
                (ind_arg5 + map0idx),   // cell_det
                (ind_arg6 + map0idx),   // cell_electric_field
                (ind_arg7 + map0idx),   // cell_connectivity
                dir_arg8,               // full_mesh_search
                dir_arg9                // dt
            );                
            
        } while ((move_status == (int)NEED_MOVE) && (map0idx < num_cells));

        if (move_status == (int)NEED_REMOVE) /*outside the mesh*/
        {  
            printf("***** INJ : need to remove particle %d\n", n);            
            particle_statuses[n] = (int)NEED_REMOVE;
        }
        else if (move_status != MOVE_DONE)
        {
            printf("****************************** move_injected_particles_to_cell__kernel returned status[%d] for particle [%d]\n", move_status, n);
        }

        // __syncthreads();
    }
}

//*************************************************************************************************
void oppic_par_loop_particle_inject__MoveToCells(
    oppic_set set,      // particles_set
    oppic_arg arg0,     // part_position,
    oppic_arg arg1,     // part_weights,
    oppic_arg arg2,     // part_cell_index,
    oppic_arg arg3,     // part_velocity,
    oppic_arg arg4,     // cell_volume,
    oppic_arg arg5,     // cell_det,
    oppic_arg arg6,     // cell_electric_field,
    oppic_arg arg7,     // cell_connectivity_map,
    oppic_arg arg8,     // particles_injected,
    oppic_arg arg9      // dt,     
    )
{ TRACE_ME;
    
    if (OP_DEBUG) printf("FEMPIC - oppic_par_loop_particle_inject__MoveToCells num_particles %d\n", set->size);

    int nargs = 10;
    oppic_arg args[nargs];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;
    args[5] = arg5;
    args[6] = arg6;
    args[7] = arg7;
    args[8] = arg8;
    args[9] = arg9;

    cutilSafeCall(hipMalloc(&(arg8.data_d), arg8.size));
    cutilSafeCall(hipMemcpy(arg8.data_d, arg8.data, arg8.size, hipMemcpyHostToDevice));
    cutilSafeCall(hipMalloc(&(arg9.data_d), arg9.size));
    cutilSafeCall(hipMemcpy(arg9.data_d, arg9.data, arg9.size, hipMemcpyHostToDevice));

    oppic_init_particle_move(set);

    int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, Device_GPU);
    if (set_size > 0) 
    {
        opDat0_MoveToCells_inject_stride_OPPIC_HOST = arg0.dat->set->size;
        opDat1_MoveToCells_inject_stride_OPPIC_HOST = arg1.dat->set->size;
        opDat3_MoveToCells_inject_stride_OPPIC_HOST = arg3.dat->set->size;
        opDat5_MoveToCells_inject_stride_OPPIC_HOST = arg5.dat->set->size;
        opDat6_MoveToCells_inject_stride_OPPIC_HOST = arg6.dat->set->size;
        opDat7_MoveToCells_inject_stride_OPPIC_HOST = arg7.map->from->size;

        hipMemcpyToSymbol(HIP_SYMBOL(opDat0_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat0_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat1_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat1_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat3_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat3_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat5_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat5_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat6_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat6_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat7_MoveToCells_inject_stride_OPPIC_CONSTANT), &opDat7_MoveToCells_inject_stride_OPPIC_HOST, sizeof(int));

        int start   = (set->size - set->diff);
        int end     = set->size;

        if (end - start > 0) 
        {
            int nthread = GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;

            oppic_cuda_injected_MoveToCells<<<nblocks, nthread>>>(
                (int *)     set->cell_index_dat->data_d,
                (double *)  arg0.data_d,                // part_position,
                (double *)  arg1.data_d,                // part_weights,
                (int *)     arg2.data_d,                // part_cell_index,
                (double *)  arg3.data_d,                // part_velocity,
                (double *)  arg4.data_d,                // cell_volume,
                (double *)  arg5.data_d,                // cell_det,
                (double *)  arg6.data_d,                // cell_electric_field,
                (int *)     arg7.data_d,                // cell_connectivity_map,
                (bool *)    arg8.data_d,                // particles_injected,
                (double *)  arg9.data_d,                // dt,
                (int *)     set->particle_statuses_d,   // mark particles as MOVE_DONE, NEED_REMOVE etc...
                start, 
                end, 
                set->cells_set->size);
        }
    }

    cutilSafeCall(hipDeviceSynchronize());

    oppic_finalize_particle_move(set);

    cutilSafeCall(hipFree(arg8.data_d));
    cutilSafeCall(hipFree(arg9.data_d));
    op_mpi_set_dirtybit_cuda(nargs, args);
    cutilSafeCall(hipDeviceSynchronize());
}

//*************************************************************************************************