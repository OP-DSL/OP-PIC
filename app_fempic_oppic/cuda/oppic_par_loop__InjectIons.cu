#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

int opDat0_InjectIons_stride_OPPIC_HOST = -1;
int opDat1_InjectIons_stride_OPPIC_HOST = -1;
int opDat2_InjectIons_stride_OPPIC_HOST = -1;
int opDat3_InjectIons_stride_OPPIC_HOST = -1;

__constant__ int opDat0_InjectIons_stride_OPPIC_CONSTANT;
__constant__ int opDat1_InjectIons_stride_OPPIC_CONSTANT;
__constant__ int opDat2_InjectIons_stride_OPPIC_CONSTANT;
__constant__ int opDat3_InjectIons_stride_OPPIC_CONSTANT;


//user function
//*************************************************************************************************
__device__ double rnd_gpu() 
{
    return 0.0; // TODO: Make this a normally distributed random number generator
}

__device__ void inject_ions__kernel_gpu(
    double *pos,
    double *vel,
    double *ef ,
    double *lc, 
    int *cell_index
)
{
    /*sample random position on the inlet face*/
    pos[0 * opDat0_InjectIons_stride_OPPIC_CONSTANT] = -0.1 + 0.2 * rnd_gpu();
    pos[1 * opDat0_InjectIons_stride_OPPIC_CONSTANT] = -0.1 + 0.2 * rnd_gpu();
    pos[2 * opDat0_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;

    /*injecting cold beam*/
    vel[0 * opDat1_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    vel[1 * opDat1_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    vel[2 * opDat1_InjectIons_stride_OPPIC_CONSTANT] = ION_VELOCITY;

    ef[0 * opDat2_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    ef[1 * opDat2_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    ef[2 * opDat2_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;

    lc[0 * opDat3_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    lc[1 * opDat3_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    lc[2 * opDat3_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;
    lc[3 * opDat3_InjectIons_stride_OPPIC_CONSTANT] = ZERO_double;

    *cell_index = 0;
}


// CUDA kernel function
//*************************************************************************************************
__global__ void oppic_cuda_InjectIons(
    double *__restrict dir_arg0,
    double *__restrict dir_arg1,
    double *__restrict dir_arg2,
    double *__restrict dir_arg3,
    int *__restrict dir_arg4,
    int start,
    int end,
    int set_size
    ) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;
        // printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
        //user-supplied kernel call
        inject_ions__kernel_gpu(
            (dir_arg0 + n),
            (dir_arg1 + n),
            (dir_arg2 + n),
            (dir_arg3 + n),
            (dir_arg4 + n)
        );
    }
}

// Issues with Random number generation made this to be run on CPU
//*************************************************************************************************
void oppic_par_loop_inject__InjectIons_gpu(
    oppic_set set,      // particles_set
    oppic_arg arg0,     // part_position,
    oppic_arg arg1,     // part_velocity,
    oppic_arg arg2,     // part_electric_field,
    oppic_arg arg3,     // part_weights,
    oppic_arg arg4      // part_cell_index,
    )
{ TRACE_ME;

    if (OP_DEBUG) printf("FEMPIC - oppic_par_loop_inject__InjectIons num_particles %d diff %d\n", set->size, set->diff);

    int nargs = 5;
    oppic_arg args[5];

    args[0] = arg0;
    args[1] = arg1;
    args[2] = arg2;
    args[3] = arg3;
    args[4] = arg4;

    int set_size = op_mpi_halo_exchanges_grouped(set, nargs, args, Device_GPU);
    if (set_size > 0) 
    {
        opDat0_InjectIons_stride_OPPIC_HOST = arg0.dat->set->size;
        opDat1_InjectIons_stride_OPPIC_HOST = arg1.dat->set->size;
        opDat2_InjectIons_stride_OPPIC_HOST = arg2.dat->set->size;
        opDat3_InjectIons_stride_OPPIC_HOST = arg3.dat->set->size;

        hipMemcpyToSymbol(HIP_SYMBOL(opDat0_InjectIons_stride_OPPIC_CONSTANT), &opDat0_InjectIons_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat1_InjectIons_stride_OPPIC_CONSTANT), &opDat1_InjectIons_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat2_InjectIons_stride_OPPIC_CONSTANT), &opDat2_InjectIons_stride_OPPIC_HOST, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(opDat3_InjectIons_stride_OPPIC_CONSTANT), &opDat3_InjectIons_stride_OPPIC_HOST, sizeof(int));

        int start = (set->size - set->diff);
        int end   = set->size;

        if (end - start > 0) 
        {
            int nthread = GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;

            oppic_cuda_InjectIons<<<nblocks, nthread>>>(
                (double *)  arg0.data_d,
                (double *)  arg1.data_d,
                (double *)  arg2.data_d,
                (double *)  arg3.data_d,
                (int *)     arg4.data_d,
                start, 
                end, 
                set->size);
        }
    }

    op_mpi_set_dirtybit_cuda(nargs, args);
    cutilSafeCall(hipDeviceSynchronize());

// oppic_download_particle_set(set);
}

//*************************************************************************************************
void inject_ions__kernel(
    double *pos,
    double *vel,
    double *ef ,
    double *lc, 
    int *cell_index
)
{
    /*sample random position on the inlet face*/
    pos[0] = -0.1 + 0.2 * rnd();
    pos[1] = -0.1 + 0.2 * rnd();
    pos[2] = 0;

    /*injecting cold beam*/
    vel[0] = 0;
    vel[1] = 0;
    vel[2] = ION_VELOCITY;

    ef[0] = 0;
    ef[1] = 0;
    ef[2] = 0;

    lc[0] = 0.0;
    lc[1] = 0.0;
    lc[2] = 0.0;
    lc[3] = 0.0;

    *cell_index = 0;
}

//*************************************************************************************************
void oppic_par_loop_inject__InjectIons(
    oppic_set set,      // particles_set
    oppic_arg arg0,     // part_position,
    oppic_arg arg1,     // part_velocity,
    oppic_arg arg2,     // part_electric_field,
    oppic_arg arg3,     // part_weights,
    oppic_arg arg4      // part_cell_index,
    )
{ TRACE_ME;
    if (OP_DEBUG) printf("FEMPIC - oppic_par_loop_inject__InjectIons num_particles %d diff %d\n", set->size, set->diff);

    oppic_download_particle_set(set);

    for (int i = (set->size - set->diff); i < set->size; i++)
    {    
        inject_ions__kernel(    
            &((double *)arg0.data)[i * arg0.dim],            // part_position,
            &((double *)arg1.data)[i * arg1.dim],            // part_velocity,
            &((double *)arg2.data)[i * arg2.dim],            // part_electric_field,
            &((double *)arg3.data)[i * arg3.dim],            // part_weights,
            &((int *)arg4.data)[i * arg4.dim]                // part_cell_index,
        );
    }

    oppic_upload_particle_set(set);
}