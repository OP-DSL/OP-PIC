#include "hip/hip_runtime.h"
/* 
BSD 3-Clause License

Copyright (c) 2022, OP-DSL

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// AUTO GENERATED CODE

__constant__ int injectIons_stride_OPP_CUDA_0;
__constant__ int injectIons_stride_OPP_CUDA_1;
__constant__ int injectIons_stride_OPP_CUDA_4;
__constant__ int injectIons_stride_OPP_CUDA_5;
__constant__ int injectIons_stride_OPP_CUDA_6;
__constant__ int injectIons_stride_OPP_CUDA_7;
__constant__ int injectIons_stride_OPP_CUDA_8;
__constant__ int injectIons_stride_OPP_CUDA_9;

int injectIons_stride_OPP_HOST_0 = -1;
int injectIons_stride_OPP_HOST_1 = -1;
int injectIons_stride_OPP_HOST_4 = -1;
int injectIons_stride_OPP_HOST_5 = -1;
int injectIons_stride_OPP_HOST_6 = -1;
int injectIons_stride_OPP_HOST_7 = -1;
int injectIons_stride_OPP_HOST_8 = -1;
int injectIons_stride_OPP_HOST_9 = -1;

//user function
//*************************************************************************************************
__device__ void inject_ions__kernel_gpu(
    double *part_pos,
    double *part_vel,
    int *part_cell_connectivity,
    int *cell_id, 
    double *cell_ef,
    double *iface_u,
    double *iface_v,
    double *iface_normal,
    double *node_pos,
    const double* dummy_part_random
)
{
    double a = dummy_part_random[0 * injectIons_stride_OPP_CUDA_9];
    double b = dummy_part_random[1 * injectIons_stride_OPP_CUDA_9];
    if ((a + b) > 1)  
    {
        a = (1 - a);
        b = (1 - b);
    }

    for (int i = 0; i < DIMENSIONS; i++) 
    {
        part_pos[i * injectIons_stride_OPP_CUDA_0] = a * iface_u[i * injectIons_stride_OPP_CUDA_5] + b * iface_v[i * injectIons_stride_OPP_CUDA_6] + node_pos[i * injectIons_stride_OPP_CUDA_8];

        part_vel[i * injectIons_stride_OPP_CUDA_1] = (iface_normal[i * injectIons_stride_OPP_CUDA_7] * CONST_ion_velocity_cuda);
        part_vel[i * injectIons_stride_OPP_CUDA_1] -= CONST_charge_cuda / CONST_mass_cuda * cell_ef[i * injectIons_stride_OPP_CUDA_4] * (0.5 * CONST_dt_cuda);

        // double v1 = (iface_normal[i * injectIons_stride_OPP_CUDA_7] * CONST_ion_velocity_cuda);
        // double v2 = (CONST_charge_cuda / CONST_mass_cuda);
        // double v3 = (0.5 * CONST_dt_cuda);
        // double v4 = (cell_ef[i * injectIons_stride_OPP_CUDA_4] * v3);
        // double v5 = (v2 * v4);

        // part_vel[i * injectIons_stride_OPP_CUDA_1] = (v1 - v5);
        
        // printf("%d %+2.20lE - %+2.20lE %+2.20lE\n - %+2.20lE %+2.20lE %+2.20lE %+2.20lE %+2.20lE\n\n", i,
        //     part_vel[i * injectIons_stride_OPP_CUDA_1],
        //     iface_normal[i * injectIons_stride_OPP_CUDA_7], cell_ef[i * injectIons_stride_OPP_CUDA_4],
        //     v1, 
        //     v2,
        //     v3, 
        //     v4,
        //     v5);
    }

    (*part_cell_connectivity) = (*cell_id);
}

// CUDA kernel function
//*************************************************************************************************
__global__ void oppic_cuda_InjectIons(
    const int *__restrict mesh_relation,
    double *__restrict dir_arg0,
    double *__restrict dir_arg1,
    int *__restrict dir_arg2,
    int *__restrict ind_arg3,
    double *__restrict ind_arg4,
    const int *__restrict opDat4Map,
    double *__restrict ind_arg5,
    double *__restrict ind_arg6,
    double *__restrict ind_arg7,
    double *__restrict ind_arg8,
    double *__restrict dir_arg9,
    int start,
    int end,
    int inj_start
    ) 
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid + start < end) 
    {
        int n = tid + start;

        const int map0idx = mesh_relation[n + inj_start]; // iface index
        const int map1idx = opDat4Map[map0idx]; // cell index

        inject_ions__kernel_gpu(
            (dir_arg0 + (n + inj_start)),
            (dir_arg1 + (n + inj_start)),
            (dir_arg2 + (n + inj_start)),
            (ind_arg3 + map0idx),
            (ind_arg4 + map1idx),
            (ind_arg5 + map0idx),
            (ind_arg6 + map0idx),
            (ind_arg7 + map0idx),
            (ind_arg8 + map0idx),
            (dir_arg9 + n)
        );
    }
}

void oppic_par_loop_inject__InjectIons(
    oppic_set set,      // particles_set
    oppic_arg arg0,     // part_position,
    oppic_arg arg1,     // part_velocity,
    oppic_arg arg2,     // part_cell_connectivity,
    oppic_arg arg3,     // iface to cell map
    oppic_arg arg4,     // cell_ef,
    oppic_arg arg5,     // iface_u,
    oppic_arg arg6,     // iface_v,
    oppic_arg arg7,     // iface_normal,
    oppic_arg arg8,     // iface_node_pos
    oppic_arg arg9      // dummy_part_random
)
{ TRACE_ME;

    if (FP_DEBUG) printf("FEMPIC - oppic_par_loop_inject__InjectIons num_particles %d diff %d\n", set->size, set->diff);

    int nargs = 10;
    oppic_arg args[nargs] = { arg0, arg1, arg2, arg3, arg4, arg5, arg6, arg7, arg8, arg9 };

    int set_size = oppic_mpi_halo_exchanges_grouped(set, nargs, args, Device_GPU);
    if (set_size > 0) 
    {
        injectIons_stride_OPP_HOST_0 = arg0.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_1 = arg1.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_4 = arg4.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_5 = arg5.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_6 = arg6.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_7 = arg7.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_8 = arg8.dat->set->set_capacity;
        injectIons_stride_OPP_HOST_9 = arg9.dat->set->set_capacity;

        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_0), &injectIons_stride_OPP_HOST_0, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_1), &injectIons_stride_OPP_HOST_1, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_4), &injectIons_stride_OPP_HOST_4, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_5), &injectIons_stride_OPP_HOST_5, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_6), &injectIons_stride_OPP_HOST_6, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_7), &injectIons_stride_OPP_HOST_7, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_8), &injectIons_stride_OPP_HOST_8, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(injectIons_stride_OPP_CUDA_9), &injectIons_stride_OPP_HOST_9, sizeof(int));

        int start     = 0;
        int end       = set->diff;
        int inj_start = (set->size - set->diff);

        if (end - start > 0) 
        {
            int nthread = GPU_THREADS_PER_BLOCK;
            int nblocks = (end - start - 1) / nthread + 1;

            oppic_cuda_InjectIons<<<nblocks, nthread>>>(
                (int *)    set->mesh_relation_dat->data_d,
                (double *) arg0.data_d,
                (double *) arg1.data_d,
                (int *)    arg2.data_d,
                (int *)    arg3.data_d,
                (double *) arg4.data_d,
                (int *)    arg4.map_data_d,
                (double *) arg5.data_d,
                (double *) arg6.data_d,
                (double *) arg7.data_d,
                (double *) arg8.data_d,
                (double *) arg9.data_d,
                start, 
                end, 
                inj_start);
        }
    }

    oppic_mpi_set_dirtybit_grouped(nargs, args, Device_GPU);
    cutilSafeCall(hipDeviceSynchronize());
}